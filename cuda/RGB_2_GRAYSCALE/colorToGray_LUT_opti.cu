#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
#include <hip/hip_runtime.h>

//@@ INSERT DEVICE CODE HERE
#define ch 3
  //2D thread organization
__global__ void rgb_to_gry_img_lut(unsigned char* rgbI, int width, int height, unsigned char* gryI, float *Lut){
 
	int w=blockIdx.x*blockDim.x+threadIdx.x;    
	int h=blockIdx.y*blockDim.y+threadIdx.y;

	if(w <width && h<height){    //boundary check
		int index= h*width+w; //indexing for the input array in 2D
    	
		int rgb_index= index*ch;    
       	unsigned char red=rgbI[rgb_index];   //getting the pixel for red in r
       	unsigned char green=rgbI[rgb_index+1]; //getting the pixel for green
       	unsigned char blue=rgbI[rgb_index+2];  //getting the pixel for blue
   	       	
		gryI[index]= (unsigned char)Lut[((red * 256 + blue) * 256) + green];
	}
}

int main(int argc, char *argv[]) {
	    
	float *h_lut_ar = (float*) malloc(256 * 256 * 256 * sizeof(float));
    int r,g,b;
    float gray;
    
    for(r=0;r<256;r++){
       	for(g=0;g<256;g++){
           	for(b=0;b<256;b++){
            	gray = 0.21*r + 0.71*g + 0.07*b;			     	
				h_lut_ar[((r * 256) + g) * 256 + b] = gray;   		
			}
   		}
  	} 
    int imgheight;
    int imgwidth;
    int imgdepth;
    float *d_lut_ar;
    const char *right_image_path = "im0.png";
    const char *right_output_image= "rightoutput_LUT.png";
    unsigned char *h_right_input_image = stbi_load(right_image_path, &imgwidth, &imgheight, &imgdepth, 3);
    unsigned char *h_right_output_image;
    unsigned char *d_right_input_image;
    unsigned char *d_right_output_image; 
  
    float execution_time;
    hipEvent_t start_time, stop_time;
    hipEventCreate(&start_time);
    hipEventCreate(&stop_time);   

    int right_image_size = imgwidth * imgheight* imgdepth * sizeof(unsigned char);
    h_right_output_image = (unsigned char*) malloc(right_image_size);
    
    //allocationg memory on the GPU for the right image
    hipMalloc((void **)&d_right_input_image,imgwidth * imgheight * imgdepth * sizeof(unsigned char));
    hipMalloc((void **)&d_right_output_image, imgwidth * imgheight * sizeof(unsigned char));
    hipMalloc((void **)&d_lut_ar, 256 * 256 * 256 * sizeof(float)); 

    //copying image data from Host to device for the right image
    hipMemcpy(d_right_input_image, h_right_input_image, right_image_size, hipMemcpyHostToDevice);
    hipMemcpy(d_lut_ar,h_lut_ar,256*256*256 * sizeof(float),hipMemcpyHostToDevice);
    
    
    int B=32;
    const dim3 blocksize(B,B,1); //declaring the number of threads
    const dim3 gridsize((imgwidth-1)/B+1,(imgheight-1)/B+1,1); //declaring the number of blocks
    
    hipEventRecord(start_time, 0);
    //calling the kernel
    for (int j=0; j<10 ; j++){
    	rgb_to_gry_img_lut<<<gridsize,blocksize>>>(d_right_input_image, imgwidth, imgheight, d_right_output_image, d_lut_ar);
    }
    hipDeviceSynchronize();
    hipEventRecord(stop_time, 0);
    hipEventSynchronize(stop_time);
    hipEventElapsedTime(&execution_time, start_time, stop_time);
    execution_time /=10.0f;
	printf("Total execution time for LUT based Color To Grayscale conversion: (ms) %f\n",execution_time);
    

    //coping output data from Device to Host for the right image
    hipMemcpy(h_right_output_image, d_right_output_image, imgwidth * imgheight, hipMemcpyDeviceToHost);
    hipMemcpy(h_lut_ar,d_lut_ar,256*256*256,hipMemcpyDeviceToHost);
    //saving the output right image
    stbi_write_png(right_output_image, imgwidth, imgheight, 1, h_right_output_image, imgwidth);
   
    
    //freeing the device memory
    hipFree(d_right_input_image);
    hipFree(d_right_output_image);
    hipFree(d_lut_ar);
  
    //freeing the host memory
    free(h_right_output_image);
    free(h_lut_ar);
    stbi_image_free(h_right_input_image);
   

  return 0;
}
