#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
#include <iostream>
#include <chrono>
#include <vector>
#include <algorithm>
#include <string>
#include <dirent.h>
#include <sys/stat.h>
#include <sys/types.h>
#include<hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <immintrin.h>  // Include header for SIMD intrinsics
#include <fstream>


// loading cuda: module load cuda11/11.0
// Compile as:
// nvcc -o lbm lbm.cu -std=c++11


using namespace std;
// define gpu and cpu kernels 

__global__ void compute_disparity(int width, int height, int block_size, int search_range, const unsigned char* left_gray, const unsigned char* right_gray, unsigned char* disparity) {



    int min_sad = INT_MAX;
    int best_offset = 0;

    // Compute the valid range of disparities for the current pixel
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x >= block_size/2 && x < width - block_size/2 && y >= block_size/2 && y < height - block_size/2) {

			int min_disp_range = max(0, x - search_range);
    	int max_disp_range = min(width - 1, x + search_range);
		  // Iterate over all possible disparities
		  for (int offset = min_disp_range - x; offset <= max_disp_range - x; offset++) {
		      // Compute SAD between left and right block
		      int sad = 0;
		      for (int i = -block_size/2; i <= block_size/2; i++) {
		          for (int j = -block_size/2; j <= block_size/2; j++) {

				            int px1 = left_gray[(y + i) * width + (x + j)];
				            int px2 = right_gray[(y + i) * width + (x + offset + j)];
				            sad += abs(px1 - px2);
								
		          }
		      }

		      // Update best disparity if current SAD is lower
		      if (sad < min_sad) {
		          min_sad = sad;
		          best_offset = offset;
		      }
		  }

		  // Store best disparity
		  disparity[(y * width) + x] = abs(best_offset); //* (255.0 / max_disparity);
		}
}


/*
__global__ void compute_disparity_optimized(int width, int height, int block_size, int search_range, const unsigned char* left_gray, const unsigned char* right_gray, unsigned char* disparity) {
    int tile_width = blockDim.x;
    int tile_height = blockDim.y;
    int tile_x = blockIdx.x * tile_width;
    int tile_y = blockIdx.y * tile_height;

    int x_start = tile_x + threadIdx.x;
    int y_start = tile_y + threadIdx.y;

    for (int y = y_start; y < tile_y + tile_height && y < height; y += blockDim.y) {

        for (int x = x_start; x < tile_x + tile_width && x < width; x += blockDim.x) {
            int min_disp_range = max(0, x - search_range);
            int max_disp_range = min(width - 1, x + search_range);

            int min_sad = INT_MAX;
            int best_offset = 0;

            for (int offset = min_disp_range - x; offset <= max_disp_range - x; offset++) {
                int sad = 0;

                for (int i = 0; i < block_size; i++) {

                    for (int j = 0; j < block_size; j++) {
                        int px1 = left_gray[(y + i) * width + (x + j)];
                        int px2 = right_gray[(y + i) * width + (x + offset + j)];
                        sad += abs(px1 - px2);
												
                    }
										if(sad >= min_sad) break;
                }

                if (sad < min_sad) {
                    min_sad = sad;
                    best_offset = offset;
                }
            }

            disparity[y * width + x] = abs(best_offset);
        }
    }
}*/


__global__ void compute_disparity_optimized(int width, int height, int block_size, int search_range, const unsigned char* left_gray, const unsigned char* right_gray, unsigned char* disparity) {
    int tile_width = blockDim.x;
    int tile_height = blockDim.y;
    int tile_x = blockIdx.x * tile_width;
    int tile_y = blockIdx.y * tile_height;

    int x_start = tile_x + threadIdx.x;
    int y_start = tile_y + threadIdx.y;

    for (int y = y_start; y < tile_y + tile_height && y < height - block_size/2 && y >=block_size/2; y += blockDim.y) {
        for (int x = x_start; x < tile_x + tile_width && x < width - block_size/2 && x >=block_size/2; x += blockDim.x) {

            int min_disp_range = max(0, x - search_range);
            int max_disp_range = min(width - 1, x + search_range);

            int min_sad = INT_MAX;
            int best_offset = 0;
						int offset = min_disp_range - x; 	
            exit: while(offset <= max_disp_range - x){
                int sad = 0;
                for (int i = -block_size/2; i <= block_size/2; i++) {
                    for (int j = -block_size/2; j <= block_size/2; j++) {
                        int px1 = left_gray[(y + i) * width + (x + j)];
                        int px2 = right_gray[(y + i) * width + (x + offset + j)];
                        sad += abs(px1 - px2);

                    }
												if(min_sad <= sad){
														offset++;
														goto exit;
												}
                }

                if (sad < min_sad) {
                    min_sad = sad;
                    best_offset = offset;
                }
								offset++;
            }

            disparity[y * width + x] = abs(best_offset);
        }
    }
}







// Function to convert RGB image to grayscale
void rgb2gray(unsigned char* rgb_img, int img_width, int img_height, unsigned char* gray_img)
{
    for (int i = 0; i < img_width * img_height; i++) {
        int r = rgb_img[3 * i];
        int g = rgb_img[3 * i + 1];
        int b = rgb_img[3 * i + 2];
        gray_img[i] = (unsigned char)(0.299 * r + 0.587 * g + 0.114 * b);
    }
}

// Function to normalize image for visualization
void normalize_image(unsigned char* img, int img_width, int img_height)
{
  	vector<unsigned char> img_vec(img, img + img_width * img_height);
    auto minmax = minmax_element(img_vec.begin(), img_vec.end());
    int min_val = *minmax.first;
    int max_val = *minmax.second;
    for (int i = 0; i < img_width * img_height; i++) {
        img[i] = (unsigned char)(255.0 * (img[i] - min_val) / (max_val - min_val));
    }
}

int main(int argc, char** argv)
{
		// Check command line arguments
		if (argc != 5) {
				cerr << "Usage: " << argv[0] << " <gpu_block_size> <cost block size> <search range> <image directory>" << endl;
				return 1;
		}


    // Parse block size and search range from command line arguments
		int gpu_block_dim = atoi(argv[1]);
    int block_size = atoi(argv[2]);
    int search_range = atoi(argv[3]);
 		string image_dir = argv[4];
   	string left_path = image_dir + "/view5.png";
   	string right_path = image_dir + "/view1.png";
    // Check that block size and search range are valid
    if (block_size <= 0 || block_size % 2 == 0 || search_range <= 0) {
        cerr << "Error: invalid block size or search range" << endl;
        return 1;
    }

    // Read input images
    int left_width, left_height, left_channels;
    unsigned char* left_data = stbi_load(left_path.c_str(), &left_width, &left_height, &left_channels, 3);
    if (!left_data) {
        cerr << "Error: could not read the left input image 'view5.png'" << endl;
        return 1;
    }

    int right_width, right_height, right_channels;
    unsigned char* right_data = stbi_load(right_path.c_str(), &right_width, &right_height, &right_channels, 3);
    if (!right_data) {
        cerr << "Error: could not read the right image 'view1.png'" << endl;
        return 1;
    }

    // Convert input images to grayscale

    unsigned char* left_gray_data = new unsigned char[left_width * left_height];
    rgb2gray(left_data, left_width, left_height, left_gray_data);

    unsigned char* right_gray_data = new unsigned char[right_width * right_height];
    rgb2gray(right_data, right_width, right_height, right_gray_data);
		// Create output_images directory if it doesn't exist
		string output_dir = image_dir + "output_images/";
		DIR* dir = opendir(output_dir.c_str());
		if (dir) {
				closedir(dir);
		} else {
				mkdir(output_dir.c_str(), 0777);
		}  

		string output_path = output_dir + "/left_gray.png";
		// Write grayscale images to disk
		if (!stbi_write_png(output_path.c_str(), left_width, left_height, 1, left_gray_data, left_width)) {
				cerr << "Error: could not write output image 'left_gray.png'" << endl;
				return 1;
		}

  	output_path = output_dir + "/right_gray.png";	

		if (!stbi_write_png(output_path.c_str(), right_width, right_height, 1, right_gray_data, right_width)) {
				cerr << "Error: could not write output image 'right_gray.png'" << endl;
				return 1;
		}

		// Allocate memory for output disparity map
		int disp_width = left_width;
		int disp_height = left_height;
		unsigned char* disparity = new unsigned char[disp_width * disp_height];
 		auto start_time = std::chrono::system_clock::now();   
		// Declare and allocate device memory
		unsigned char *d_left_gray, *d_right_gray, *d_disparity;
		hipMalloc((void **)&d_left_gray, left_width * left_height * sizeof(unsigned char));
		hipMalloc((void **)&d_right_gray, right_width * right_height * sizeof(unsigned char));
		hipMalloc((void **)&d_disparity, disp_width * disp_height * sizeof(unsigned char));

		// Copy input data to device
		hipMemcpy(d_left_gray, left_gray_data, left_width * left_height * sizeof(unsigned char), hipMemcpyHostToDevice);
		hipMemcpy(d_right_gray, right_gray_data, right_width * right_height * sizeof(unsigned char), hipMemcpyHostToDevice);

		// Set up kernel grid and block size
		//int gpu_block_dim = 2;
		dim3 block(gpu_block_dim,gpu_block_dim, 1);
		dim3 grid(ceil(disp_width + gpu_block_dim - 1) / gpu_block_dim, ceil(disp_height + gpu_block_dim - 1) / gpu_block_dim, 1);

		float elapsed_time = 0.0f;
		for (int i = 0; i < 5; i++) {
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);
				hipEventRecord(start);

				//compute_disparity<<<grid, block>>>(disp_width, disp_height, block_size, search_range, d_left_gray, d_right_gray, d_disparity);
				compute_disparity_optimized<<<grid, block>>>(disp_width, disp_height, block_size, search_range, d_left_gray, d_right_gray, d_disparity);
				hipEventRecord(stop);
				hipEventSynchronize(stop);

				float time_i = 0.0f;
				hipMemcpy(disparity, d_disparity, disp_width * disp_height * sizeof(unsigned char), hipMemcpyDeviceToHost);
				hipEventElapsedTime(&time_i, start, stop);
				elapsed_time += time_i;
		}
		float avg_time = elapsed_time / 10.0f;
    // Write the input configurations and elapsed time to CSV file
    ofstream outfile;
    outfile.open("results.csv", ios_base::app); // open the file in append mode
    outfile << gpu_block_dim << "," << block_size << "," << search_range << "," << image_dir << "," << avg_time << endl;
    outfile.close();
    
    cout << "Average disparity estimation kernel execution time: " << avg_time << "ms" << endl;


	
		auto end_time = std::chrono::system_clock::now();
		auto program_time = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
		cout << "Total execution time: " << program_time << "ms" << endl;
		// Normalize output image for visualization
		normalize_image(disparity, disp_width, disp_height);





		output_path = output_dir + "/disparity_gpu_" + to_string(block_size) + "_" + to_string(search_range)  + "_.png";

		// Write output disparity map to file
		stbi_write_png(output_path.c_str(), disp_width, disp_height, 1, disparity, disp_width);

		// Free device and host memory
		hipFree(d_left_gray);
		hipFree(d_right_gray);
		hipFree(d_disparity);
		delete[] left_data;
		delete[] right_data;
		delete[] left_gray_data;
		delete[] right_gray_data;
		delete[] disparity;
	

		return 0;    
	
}
		  

